#include <stdio.h>
# include "hip/hip_runtime.h"
# include "hip/hip_runtime_api.h"


__global__
void add(int n, float *x, float *y, float *z)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) z[i] = x[i] + y[i];
  if (i< n) z[i]++;
}

int main() {
    int N = 1<<10;
    float *x, *y, *z, *d_x, *d_y, *d_z;

    hipDeviceReset();
    //Allocating memory onto host
    hipHostAlloc((void **)&x,  N*sizeof(float), hipHostMallocMapped );
    hipHostAlloc((void **)&y,  N*sizeof(float), hipHostMallocMapped );
    hipHostAlloc((void **)&z,  N*sizeof(float), hipHostMallocMapped );
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    //Getting device pointer
    hipHostGetDevicePointer((void **)&d_x, x, 0);
    hipHostGetDevicePointer((void **)&d_y, y, 0);
    hipHostGetDevicePointer((void **)&d_z, z, 0);

    hipDeviceSynchronize();
    hipProfilerStart();
    add<<<(N+255)/256, 256>>>(N, d_x, d_y, d_z);
    hipProfilerStop();
    hipDeviceSynchronize();

    float sum = 0.0;
    for (int i=0; i<N; i++) {
        sum = z[i] + sum;
    }

    printf("Sum = %f\n", sum);
    hipHostFree(x);
    hipHostFree(y);
    hipHostFree(z);
    hipDeviceReset();
}
